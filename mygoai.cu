#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "board.h"

struct GameBoard;
void host_clear_visited(GameBoard* this_board);
void host_board_construct(GameBoard* this_board, int s);
int host_board_addStone(GameBoard* this_board, int row, int col, int state);
void host_delete_stone(GameBoard* this_board, int row, int col);
int host_get_liberties(GameBoard* this_board, int row, int col);
int host_checkStone(GameBoard* this_board, int row, int col, int state);

inline void cleanBoard(GameBoard* myboard){
	for (int i = 0; i < 361; ++i)
		myboard->visited[i] = 0;
	return;
}

void buildBoard(GameBoard* myboard, int size){
	myboard->size = size;
	//for a new game, always let black play first
	myboard->current_player_state = BLACK;
	myboard->last_move = WHITE;
	for (int i = 0; i < size; ++i){
		for (int j = 0; j < size; ++j){	
			//initialize if this a stone or not
			myboard->draw[i*size+j] = 0;
			myboard->eval[i*size+j] = 0;
			myboard->classify[i*size+j] = 0;
		}
	}
	return;
}

int addStone(GameBoard* myboard, int row, int col, int state){
	//check boundary
	if (row < 0 || row >= myboard->size || col < 0 || col >= myboard->size )
		return 0;
	int size = myboard->size;
	//stone exist
	if(myboard->draw[row*size+col]) return 0; 
	//first put the stone, then check the state
	myboard->draw[row*size+col] = state;

	if (!checkStone(myboard, row, col, state)){
		//get removed
		myboard->draw[row*size+col] = 0;
		return 0;
	}
	//myboard->last_move = row*size+col;
	//getTerr(myboard);
	//for test purpose
	//printClasearchify(myboard);
	return 1;
}

void deleteStone(GameBoard* myboard, int row, int col){
	//recursively traverse
	int size = myboard->size;
	if (myboard->visited[row*size+col])
		return;
	myboard->visited[row*size+col] = 1;
	int state = myboard->visited[row*size+col];
	//make sure not hit the boundary
	if (row > 0){	
		if (myboard->draw[(row-1)*size+col] == state) {
			deleteStone(myboard, row-1, col);
			myboard->draw[(row-1)*size+col] = 0;
		}
	}	
	if(row < size - 1){
		if(myboard->draw[(row+1)*size+col] == state){
			deleteStone(myboard, row+1, col);
			myboard->draw[(row+1)*size+col] = 0;
		}
	}
	if (col>0){
		if (myboard->draw[row*size+col-1] == state){
			deleteStone(myboard, row, col-1);
			myboard->draw[row*size+col-1] = 0;
		}
	}
	if (col<size-1){
		if (myboard->draw[row*size+col+1] == state){
			deleteStone(myboard, row, col+1);
			myboard->draw[row*size+col+1] = 0;
		}
	}
	myboard->draw[row*size+col] = 0;
	return;
}

int countLiberty(GameBoard* myboard, int row, int col){
	int size = myboard->size;
	int count = 0;
	//check boundary
	if (myboard->visited[row*size+col] == 1)
		return 0;
	myboard->visited[row*size+col] = 1;
	int state = myboard->draw[row*size+col];
	if (row > 0){	
		if (myboard->draw[(row-1)*size+col] == state) 
			count += countLiberty(myboard,row-1,col);
		//same color or no opponent stone
		else count += (myboard->draw[(row-1)*size+col] == 0);
	}
	if (row < size-1){	
		if (myboard->draw[(row+1)*size+col] == state) 
			count += countLiberty(myboard,row+1,col);
		//same color or no opponent stone
		else count += (myboard->draw[(row+1)*size+col] == 0);
	}
	if (col > 0){	
		if (myboard->draw[row*size+col-1] == state) 
			count += countLiberty(myboard,row,col-1);
		//same color or no opponent stone
		else count += (myboard->draw[row*size+col-1] == 0);
	}
	if (col < size-1){	
		if (myboard->draw[row*size+col+1] == state) 
			count += countLiberty(myboard,row,col+1);
		//same color or no opponent stone
		else count += (myboard->draw[row*size+col+1] == 0);
	}

	return count;
}

int checkStone(GameBoard* myboard, int row, int col, int state){
	int neighbors[4];
	int size = myboard->size;
	//set the boundary
	if (row > 0) neighbors[0] = (row-1)*size +col; 
	else neighbors[0] = -1;
	if (row < size-1) neighbors[1] = (row+1)*size +col; 
	else neighbors[1] = -1;
	if (col > 0) neighbors[2] = row*size +col-1; 
	else neighbors[2] = -1;	 
	if (col < size-1) neighbors[3] = row*size +col+1; 
	else neighbors[3] = -1;	 

	int flag = 1;
	//dead
	if (!countLiberty(myboard, row, col))
		flag = 0;
	int indr,indc;
	for (int i = 0; i < 4; ++i)
	{
		if (neighbors[i] != -1 && myboard->draw[neighbors[i]] == -state){
			indr = neighbors[i]/size;
			indc = neighbors[i]%size;
			cleanBoard(myboard);
			//dead
			if (!countLiberty(myboard, indr, indc)){
				cleanBoard(myboard);
				deleteStone(myboard, indr, indc);
				flag = 1;
			}
		}
	}
	return flag;
}


__global__ void
kernel_monte_carlo(int* stones, int s, int* result){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (stones[index * s * s] != -2){
        int eval[361];
        for (int i=0; i< s*s; i++){
            eval[i] = 0;
        }

        int idx, dist, diff;
        //calculate eval
        for (int r = 0; r < s; r++){
            for (int c = 0; c < s; c++){
                idx = r * s + c;
                if (stones[idx + index * s * s] != 0){
                    diff = stones[idx + index * s * s];

                    int i1,i2,j1,j2;

                    if (r-4 > 0){i1 = r-4;} else {i1=0;}
                    if (r+5 < s){i2 = r+5;} else {i2=s;}
                    if (c-4 > 0){j1 = c-4;} else {j1=0;}
                    if (c+5 < s){j2 = c+5;} else {j2=s;}

                    for(int i = i1; i < i2; i++){
                        for(int j = j1; j < j2; j++){

                            int ab1, ab2,m;

                            if (r-i > 0){ ab1 = r-i;}
                            else {ab1 = i-r;}
                            if (c-j > 0){ ab2 = c-j;}
                            else {ab2 = j-c;}

                            if(dist == 4) m = 1;
                            if(dist == 3) m = 2;
                            if(dist == 2) m = 4;
                            if(dist == 1) m = 8;
                            if(dist == 0) m = 16;
                            if(dist > 4)  m = 0;

                            eval[i * s + j] += diff * m;
                        }
                    }
                }
            }
        }

        int w_count = 0;
        for(int i = 0; i < s * s; i++) {
            if(stones[i + index * s * s] == 1) {
                if(eval[i] < 0) w_count += 1;
                else w_count -= 1;
            }
            else if(stones[i + index * s * s] == -1) {
                if(eval[i] > 0) w_count -= 1;
                else w_count += 1;
            }
            else if(eval[i] > 0) w_count -= 1;
            else if(eval[i] < 0) w_count += 1;
        }
        result[index] = w_count;
    }
}

static inline int raisePwr(int num, int times){
	int pwr = 1;
	for (int i = 0; i < times; ++i)
		pwr *= num;
	return pwr;
}
int Monte_Carlo_Cuda(GameBoard* this_board, int n) {
    int s = this_board->size;
    int ss = s;
    if (n == 2 and s == 19) ss = 8;
    if (n == 3 and s == 9) ss = 5;
    if (n == 3 and s == 19) ss = 4;
    int num = raisePwr(ss, 2*n);
    int partial_num = int(num / (ss * ss));

    const int threadsPerBlock = 128;
    const int blocks = (num + threadsPerBlock - 1) / threadsPerBlock;

    int result[num];
    for(int i = 0; i < num; i++) result[i] = 100;

    int stones[num * s * s];
    int move_seq[num * n];

    //generating moving sequences

    int startx = 0;
    int starty = 0;
    int last_row = this_board->last_move / s;
    int last_col = this_board->last_move % s;
    if (last_row + int(ss / 2) >= s){startx = s - ss;}
    else if (last_row - int(ss / 2) > 0) {startx = last_row - int(ss/2);}

    if (last_col + int(ss / 2) >= s){starty = s - ss;}
    else if (last_col - int(ss / 2) > 0) {starty = last_col - int(ss/2);}

    int p = 0;
    //printf("startx = %d, starty = %d\n", startx, starty);
    for (int i=startx; i<startx + ss; i++){
        for (int j=starty; j<starty + ss; j++){
            for (int k=1; k<partial_num; k++){
                move_seq[p * n] = i * s + j;
                p += 1;    
            }
        }
    }

    for (int i=0; i<num * n; i++){
        if (i % n != 0) move_seq[i] = rand() % (s * s);
    }

    for (int idx = 0; idx < num; idx ++){
        GameBoard* next_board = new GameBoard;
        buildBoard(next_board, s);
        for (int r = 0; r < s; r++){
            for (int c = 0; c < s; c++){
                next_board->draw[r * s + c] = this_board->draw[r * s + c];
            }
        }

        int flag = 1;
        int type = 1;
        int cur_flag;
        for (int k=0; k<n; k++){
            type *= (-1);
            cur_flag = addStone(next_board, move_seq[idx * n + k] / s, move_seq[idx * n + k] % s, -1);
            if (cur_flag == 0){
                flag = 0;
                break;
            }
        }

        if (flag == 1){
            for (int i = 0; i < s * s; i++){
                stones[idx * s * s + i] = next_board->draw[i];
            }
        } else {
            stones[idx * s * s] = -2;
        }
        delete next_board;
    }

    int* device_stones;
    int* device_result; 

    hipMalloc(&device_stones, num * s * s * sizeof(int));
    hipMemcpy(device_stones, stones, num * s * s * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&device_result, num * sizeof(int));
    hipMemcpy(device_result, result, num * sizeof(int), hipMemcpyHostToDevice);

    // size_t limit = 0;
    // hipDeviceGetLimit(&limit, hipLimitStackSize);
    // printf("stack size = %u\n", (unsigned)limit);
    // limit = 65535;
    // hipDeviceSetLimit(hipLimitStackSize, limit);
    // hipDeviceGetLimit(&limit, hipLimitStackSize);
    // printf("stack size = %u\n", (unsigned)limit);

    kernel_monte_carlo<<<blocks, threadsPerBlock>>>(device_stones, s, device_result);
    hipDeviceSynchronize();

    hipMemcpy(result, device_result, num * sizeof(int), hipMemcpyDeviceToHost);
    // for (int i=0; i < num; i++){
    //     printf("(%d, %d) -> %d\n", move_seq[i * n], move_seq[i*n+1], result[i]);
    // }
    // printf("\n");

    // for (int idx = 0; idx < num; idx ++){
    //     if (result[idx] > max_val){
    //         max_val = result[idx];
    //         max_pos = move_seq[idx * n];
    //     }
    // }

    int max_pos = rand() % (s * s);
    float max_val = -101.0;
    int local_sum = 0;
    int local_cnt = 0;
    for (int idx = 0; idx < ss * ss; idx++){
        local_sum = 0;
        local_cnt = 0;
        for (int i=0; i < partial_num; i++){
            if (result[idx * partial_num + i] != 100){
                local_cnt += 1;
                local_sum += result[idx * partial_num + i];
            }
        }

        if (float(local_sum) / local_cnt > max_val){
            max_val = float(local_sum) / local_cnt;
            max_pos = move_seq[idx * partial_num];
        }
    }
    hipFree(result);
    hipFree(device_stones);
    hipFree(device_result);

    return max_pos;
}
